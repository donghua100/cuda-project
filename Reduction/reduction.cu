#include <hip/hip_runtime.h>
#include <stdio.h>

#define blockThreadNum 512
template<typename T, size_t N>
__global__ void reduce_basic(T *in, T *out) {
    __shared__ T sdata[blockThreadNum];
    int tx = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) sdata[tx] = in[idx];
    else sdata[tx] = 0;
    __syncthreads();

    for (int s = 1; s < blockThreadNum; s *= 2) {
        if (tx % (2*s) == 0) {
            sdata[tx] += sdata[tx + s];
        }
        __syncthreads();
    }
    if (tx == 0) out[blockIdx.x] = sdata[0];

}

template<typename T, size_t N>
__global__ void reduce1(T *in, T *out) {
    __shared__ T sdata[blockThreadNum];
    int tx = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) sdata[tx] = in[idx];
    else sdata[tx] = 0;
    __syncthreads();

    for (int s = 1; s < blockThreadNum; s *= 2) {
        idx = 2*s*tx;
        if (idx < blockThreadNum) {
            sdata[idx] += sdata[idx + s];
        }
        __syncthreads();
    }
    if (tx == 0) out[blockIdx.x] = sdata[0];
}


template<typename T, size_t N>
__global__ void reduce2(T *in, T *out) {
    __shared__ T sdata[blockThreadNum];
    int tx = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) sdata[tx] = in[idx];
    else sdata[tx] = 0;
    __syncthreads();

    for (int s = blockThreadNum / 2; s > 0; s >>= 1) {
        if (tx < s) {
            sdata[tx] += sdata[tx + s];
        }
        __syncthreads();
    }
    if (tx == 0) out[blockIdx.x] = sdata[0];
}

template<typename T, size_t N>
__global__ void reduce3(T *in, T *out) {
    __shared__ T sdata[blockThreadNum];
    int tx = threadIdx.x;
    int i = blockIdx.x*(2*blockDim.x) + threadIdx.x;
    if (i < N) sdata[tx] = in[i];
    else sdata[tx] = 0;
    if (i + blockDim.x < N) sdata[tx] += in[i + blockDim.x];
    __syncthreads();
    for (int s = blockThreadNum/2; s > 0; s >>= 1) {
        if (tx < s) {
            sdata[tx] += sdata[tx + s];
        }
        __syncthreads();
    }
    if (tx == 0) out[blockIdx.x] = sdata[0];
}

template<typename T>
__device__ void warpReduce(volatile T *sdata, int tx) {
    sdata[tx] += sdata[tx + 32];
    sdata[tx] += sdata[tx + 16];
    sdata[tx] += sdata[tx + 8];
    sdata[tx] += sdata[tx + 4];
    sdata[tx] += sdata[tx + 2];
    sdata[tx] += sdata[tx + 1];
}
template<typename T, size_t N>
__global__ void reduce4(T *in, T *out) {
    __shared__ T sdata[blockThreadNum];
    int tx = threadIdx.x;
    int i = blockIdx.x*(2*blockDim.x) + threadIdx.x;
    if (i < N) sdata[tx] = in[i];
    else sdata[tx] = 0;
    if (i + blockDim.x < N) sdata[tx] += in[i + blockDim.x];
    __syncthreads();
    for (int s = blockThreadNum/2; s > 32; s >>= 1) {
        if (tx < s) {
            sdata[tx] += sdata[tx + s];
        }
        __syncthreads();
    }
    if (tx < 32) {
        warpReduce<T>(sdata, tx);
    }
    if (tx == 0) out[blockIdx.x] = sdata[0];
}
template<typename T, size_t blocksize>
__device__ void warpReduce2(volatile  T *sdata, int tx) {
    if (blocksize >= 64) sdata[tx] += sdata[tx + 32];
    if (blocksize >= 32) sdata[tx] += sdata[tx + 16];
    if (blocksize >= 16) sdata[tx] += sdata[tx + 8];
    if (blocksize >= 8) sdata[tx] += sdata[tx + 4];
    if (blocksize >= 4) sdata[tx] += sdata[tx + 2];
    if (blocksize >= 2) sdata[tx] += sdata[tx + 1];
}

template<typename T, size_t N, size_t blocksize>
__global__ void reduce5(T *in, T *out) {
    __shared__ T sdata[blockThreadNum];
    int tx = threadIdx.x;
    int i = blockIdx.x * (2*blockDim.x) + threadIdx.x;
    if (i < N) sdata[tx] = in[i];
    else sdata[tx] = 0;
    if (i + blockDim.x < N) sdata[tx] += in[i + blockDim.x];
    __syncthreads();
    if (blocksize >= 1024) {
        if (tx < 512) sdata[tx] += sdata[tx + 512];
        __syncthreads();
    }
    if (blocksize >= 512) {
        if (tx < 256) sdata[tx] += sdata[tx + 256];
        __syncthreads();
    }
    if (blocksize >= 256) {
        if (tx < 128) sdata[tx] += sdata[tx + 128];
        __syncthreads();
    }
    if (blocksize >= 128) {
        if (tx < 64) sdata[tx] += sdata[tx + 64];
        __syncthreads();
    }
    if (tx < 32) warpReduce2<T, blocksize>(sdata, tx);
    if (tx == 0) out[blockIdx.x] = sdata[0];
}

template<typename T, size_t n>
void rand_arr(T *a) {
    for (int i = 0; i < n; i++) {
        a[i] = rand() % 10;
    }
}

template<typename T, size_t n>
void one_arr(T *a) {
    for (int i = 0; i < n; i++) {
        a[i] = 1;
    }
}

template<typename T, size_t n>
void print_arr(T *a) {
    for (int i = 0; i < n; i++) {
        printf("%d ", a[i]);
    }
    printf("\n");
}
#define CUDA_CHECK(status) \
do { \
    hipError_t cuda_status = (status); \
    if (cuda_status != hipSuccess) { \
        printf("cuda error encountered at line %d: %s\n", __LINE__, hipGetErrorString(cuda_status)); \
    } \
} while(0)

int main() {
    #define N 1024000
    typedef int T;
    #define gridblock  ((N - 1)/blockThreadNum + 1)
    T *h_in, *h_out;
    T *d_in, *d_out;
    h_in = (T *)malloc(sizeof(T)*N);
    h_out = (T *)malloc(sizeof(T)*gridblock);

    rand_arr<T, N>(h_in);
    one_arr<T, N>(h_in);
    T s = 0;
    for (int i = 0; i < N; i++) s += h_in[i];
    CUDA_CHECK(hipMalloc((void **)&d_in, sizeof(T)*N));
    CUDA_CHECK(hipMalloc((void **)&d_out, sizeof(T)*gridblock));
    CUDA_CHECK(hipMemcpy(d_in, h_in, sizeof(T)*N, hipMemcpyHostToDevice));


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    reduce_basic<T, N><<<gridblock, blockThreadNum>>>(d_in, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float millseconds;
    hipEventElapsedTime(&millseconds, start, stop);
    // printf("Time %.2f ms,  %.2f GFLOPS\n",millseconds,  N*(1e-6)/millseconds);
    printf("Time %.2f ms,  %.2f GB/s\n",millseconds,  N*sizeof(T)*1000.0/1024/1024/1024/millseconds);
    CUDA_CHECK(hipMemcpy(h_out, d_out, sizeof(T)*gridblock, hipMemcpyDeviceToHost));


    hipEventRecord(start);
    reduce1<T, N><<<gridblock, blockThreadNum>>>(d_in, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millseconds, start, stop);
    printf("Time %.2f ms,  %.2f GB/s\n",millseconds,  N*sizeof(T)*1000.0/1024/1024/1024/millseconds);
    CUDA_CHECK(hipMemcpy(h_out, d_out, sizeof(T)*gridblock, hipMemcpyDeviceToHost));

    hipEventRecord(start);
    reduce2<T, N><<<gridblock, blockThreadNum>>>(d_in, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millseconds, start, stop);
    printf("Time %.2f ms,  %.2f GB/s\n",millseconds,  N*sizeof(T)*1000.0/1024/1024/1024/millseconds);
    CUDA_CHECK(hipMemcpy(h_out, d_out, sizeof(T)*gridblock, hipMemcpyDeviceToHost));

    hipEventRecord(start);
    // #define blockThreadNums (blockThreadNum/2)
    #define gridblocks (gridblock/2)
    reduce3<T, N><<<gridblocks, blockThreadNum>>>(d_in, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millseconds, start, stop);
    printf("Time %.2f ms,  %.2f GB/s\n",millseconds,  N*sizeof(T)*1000.0/1024/1024/1024/millseconds);
    CUDA_CHECK(hipMemcpy(h_out, d_out, sizeof(T)*gridblock, hipMemcpyDeviceToHost));

    hipEventRecord(start);
    reduce4<T, N><<<gridblocks, blockThreadNum>>>(d_in, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millseconds, start, stop);
    printf("Time %.2f ms,  %.2f GB/s\n",millseconds,  N*sizeof(T)*1000.0/1024/1024/1024/millseconds);
    CUDA_CHECK(hipMemcpy(h_out, d_out, sizeof(T)*gridblock, hipMemcpyDeviceToHost));

    hipEventRecord(start);
    reduce5<T, N, blockThreadNum><<<gridblocks, blockThreadNum>>>(d_in, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&millseconds, start, stop);
    printf("Time %.2f ms,  %.2f GB/s\n",millseconds,  N*sizeof(T)*1000.0/1024/1024/1024/millseconds);
    CUDA_CHECK(hipMemcpy(h_out, d_out, sizeof(T)*gridblock, hipMemcpyDeviceToHost));

    T ss = 0;
    for (int i = 0; i < gridblocks; i++) ss += h_out[i];
    printf("%d %d\n", s, ss);
    // print_arr<T, gridblock>(h_out);

    return 0;
}
